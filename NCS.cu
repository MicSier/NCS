#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <sstream>
#include <chrono>
#include <numeric>
#include <vector>
#include "cmath"
#include <string>
#include <cmath>
#include <hip/hip_runtime.h>
#include <string>
#include <fstream>

using namespace std;

#define BLOCK_SIZE 16
const double h_pi = 3.141592653597932384;
__constant__ double pi_device = 3.141592653597932384;
const double pi = 3.141592653597932384;
const double delta = 0.0000000001;
const int n = 10000, k = 5;

const double xk[] = { 0.9061798459386641,-0.9061798459386641,0.538469310105683,-0.5384693101056829,0.0 };
const double ak[] = { 0.2369268850561876,0.2369268850561876,0.47862867049936647,0.47862867049936586,0.5688888888888889 };

typedef double (*Under_Integral_Func)(double, double, double, double, double);

__device__ double g(double x, double gamma0)
{
	return gamma0 * (abs(sin(x)));
}

__device__ double ek(double x, double tp)
{
	return -2.0 * (cos(x)) + 2.0 * tp * cos(2 * x);
}

__device__ double sc_uifunc(double x, double Tc, double mi, double gamma0, double tp)
{
	double ksip = ek(x, tp) - mi + g(x, gamma0), ksim = ek(x, tp) - mi - g(x, gamma0);

	return tanh(ksip / (2.0 * Tc)) + tanh(ksim / (2.0 * Tc));
}

__device__ double singlet_uifunt(double x, double Tc, double mi, double gamma0, double tp)
{
	double ksip = ek(x, tp) - mi + g(x, gamma0), ksim = ek(x, tp) - mi - g(x, gamma0);
	double res;

	if (abs(ksim) < 0.0000000000001)
		res = 1.0 / (2.0 * Tc);
	else
		res = tanh(ksim / (2.0 * Tc)) / (2.0 * ksim);

	if (abs(ksip) < 0.0000000000001)
		res += 1.0 / (2.0 * Tc);
	else
		res += tanh(ksip / (2.0 * Tc)) / (2.0 * ksip);

	return res;
}

__device__ double gd(double x, double gamma0)
{
	return gamma0 * (sin(x) * sin(x));
}

__device__ double d2(double x)
{
	return pow(sin(x), 2.0);
}

__device__ double tryplet_uifunt(double x, double Tc, double mi, double gamma0, double tp)
{
	double ksi = ek(x, tp) - mi, eps = 0.000000001, vg = g(x, gamma0);
	double uifunt1 = 0.0, uifunt2 = 0.0, pare = 0.0, uifunt;
	if ((abs(ksi) < eps) && (abs(vg) < eps))
	{
		uifunt1 = 1.0 / (2.0 * Tc);
	}
	else
	{
		if (abs(ksi) < eps)
		{
			uifunt1 = tanh(vg / (2.0 * Tc)) / vg;
			pare = d2(x) - pow(gd(x, gamma0) / vg, 2.0);

			if (abs(vg / Tc) > 100)
			{
				uifunt2 = -1.0 / vg;
			}
			else if (abs(vg / Tc) < eps)
			{
				uifunt2 = 0.0;
			}
			else
			{
				uifunt2 = 1.0 / (Tc + Tc * cosh(vg / Tc)) - sinh(vg / Tc) / (vg + vg * cosh(vg / Tc));
			}

		}
		else
		{
			if (abs(vg) < eps)
			{
				uifunt1 = tanh((ksi) / (2.0 * Tc)) / (ksi);
			}
			else if ((abs(ksi + vg) < eps) || (abs(ksi - vg) < eps))
			{
				uifunt1 = (Tc * tanh(vg / Tc) + vg) / (4.0 * Tc * vg);
				uifunt2 = 0.250 * (tanh(vg / Tc) / vg - 1.0 / Tc);
				pare = d2(x) - pow(gd(x, gamma0) / vg, 2.0);
			}
			else
			{
				uifunt1 = tanh((ksi + vg) / (2.0 * Tc)) / (2.0 * (ksi + vg)) + tanh((ksi - vg) / (2.0 * Tc)) / (2.0 * (ksi - vg));
				uifunt2 = vg / ksi * (tanh((ksi + vg) / (2.0 * Tc)) / (2.0 * (ksi + vg)) - tanh((ksi - vg) / (2.0 * Tc)) / (2.0 * (ksi - vg)));
				pare = d2(x) - pow(gd(x, gamma0) / vg, 2.0);
			}

		}

	}
	uifunt = uifunt1 * d2(x) + uifunt2 * pare;
	return uifunt;
}

__device__ Under_Integral_Func uif_array[] = { sc_uifunc, singlet_uifunt, tryplet_uifunt };
enum UI_Func_Index { i_sc_uifunc = 0, i_singlet_uifunt = 1, i_tryplet_uifunt };

__global__ void integrateKernel(double* result, UI_Func_Index i_f, double a, double h, int n, double* xk, double* ak, double Tc, double mi, double gamma0, double tp) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < n) {
		int j = idx % 5;
		int i = idx / 5;
		double z = ((2.0 * a + h * (2.0 * i + 1.0)) - h * xk[j]) * 0.5;
		result[idx] = ak[j] * ((uif_array[i_f])(z, Tc, mi, gamma0, tp));
	}
}

double sc_integrate1D_gl_gpu(UI_Func_Index i_f, double a, double b, double Tc, double mi, double gamma0, double tp) {
	double h = (b - a) / (n);
	// Host variables
	double* h_result = new double[n * k];

	// Device variables
	double* d_result;
	double* d_xk;
	double* d_ak;
	hipMalloc((void**)&d_result, n * k * sizeof(double));
	hipMalloc((void**)&d_xk, k * sizeof(double));
	hipMalloc((void**)&d_ak, k * sizeof(double));

	// Copy data to device
	hipMemcpy(d_xk, xk, k * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_ak, ak, k * sizeof(double), hipMemcpyHostToDevice);

	// Launch the kernel
	int num_blocks = (n * k + BLOCK_SIZE - 1) / BLOCK_SIZE;
	integrateKernel << <num_blocks, BLOCK_SIZE >> > (d_result, i_f, a, h, n * k, d_xk, d_ak, Tc, mi, gamma0, tp);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize failed: %s\n", hipGetErrorString(cudaStatus));
	}
	// Copy the result back to host
	hipMemcpy(h_result, d_result, n * k * sizeof(double), hipMemcpyDeviceToHost);

	double result = accumulate(h_result, h_result + n * k, 0.0) * h * 0.5;
	// Cleanup
	hipFree(d_result);
	hipFree(d_xk);
	hipFree(d_ak);
	delete[] h_result;
	return result;
}

double singlet_gap(double Tc, double mi, double gamma0, double Vs, double tp)
{
	return 1.0 - Vs / (2.0 * pi) * sc_integrate1D_gl_gpu(i_singlet_uifunt, 0.0, pi, Tc, mi, gamma0, tp);
}

double sc_occ(double mi, double Tc, double gamma0, double nl, double tp)
{
	return 1.0 - nl - sc_integrate1D_gl_gpu(i_sc_uifunc, 0.0, pi, Tc, mi, gamma0, tp) / (2.0 * pi);
}

typedef double (*Gap_Func)(double, double, double, double, double);


double occ_solve1D_zbr(Gap_Func f, double a, double b, double tol, double T, double gamma0, double nl, double tp)
{
	int itmax = 100;
	double d, r, s, e, p, q, xm, tol1, c, fa, fb, fc;

	fa = f(a, T, gamma0, nl, tp);
	fb = f(b, T, gamma0, nl, tp);

	if (fa * fb > 0.0)
	{
		cout << "occ_zbr err:Takie same znaki!  fa  " << fa << " fb  " << fb << " tc,g0,a,b  " << T << "  " << gamma0 << "  " << a << "  " << b << endl;
		return 0.0;
	}
	c = b;
	fc = fb;
	for (int i = 1; i < itmax; i++)
	{
		if (fb * fc > 0.0)
		{
			c = a;
			fc = fa;
			d = b - a;
			e = d;
		}
		if (abs(fc) < abs(fb))
		{
			a = b;
			b = c;
			c = a;
			fa = fb;
			fb = fc;
			fc = fa;
		}
		tol1 = 2.0 * delta * abs(b) + 0.5 * tol;
		xm = 0.5 * (c - b);
		if ((abs(xm) < tol1) || (fb == 0)) return b;
		if ((abs(e) > tol1) && (abs(fa) > abs(fb)))
		{
			s = fb / fa;
			if (a == c)
			{
				p = 2.0 * xm * s;
				q = 1.0 - s;
			}
			else
			{
				q = fa / fc;
				r = fb / fc;
				p = s * (2.0 * xm * q * (q - r) - (b - a) * (r - 1.0));
				q = (q - 1.0) * (r - 1.0) * (s - 1.0);
			}
			if (p > 0.0) q = -q;
			p = abs(p);
			if (2.0 * p < min(3.0 * xm * q - abs(tol1 * q), abs(e * q)))
			{
				e = d;
				d = p / q;
			}
			else
			{
				d = xm;
				e = d;
			}
		}
		else
		{
			d = xm;
			e = d;
		}
		a = b;
		fa = fb;
		if (abs(d) > tol1) b += d;
		else
		{
			if (xm > 0.0) b = b + abs(tol1);
			else b = b - abs(tol1);
		}
		fb = f(b, T, gamma0, nl, tp);


	}
	cout << "zbr exeding max iteractions!" << endl;
	return b;
}

struct Result_Pair {
	double T;
	double mi;
};

Result_Pair sc_solve1D_zbr(Gap_Func f, double a, double b, double tol, double gamma0, double Vs, double sxc, double syc, double nl, double tp)
{
	int itmax = 100;
	double d, r, s, e, p, q, xm, tol1, c, fa, fb, fc, eps = 3.0e-8, mi;
	mi = occ_solve1D_zbr(sc_occ, sxc, syc, tol, a, gamma0, nl, tp);
	fa = f(a, mi, gamma0, Vs, tp);
	mi = occ_solve1D_zbr(sc_occ, sxc, syc, tol, b, gamma0, nl, tp);
	fb = f(b, mi, gamma0, Vs, tp);
	Result_Pair res = { b, mi };

	if (fa * fb > 0.0)
	{
		cout << "sc_zbr err:Takie same znaki!  fa  " << fa << " fb  " << fb << " mi,g0,a,b  " << mi << "  " << gamma0 << "  " << a << "  " << b << endl;
		return { 0.0,0.0 };
	}
	c = b;
	fc = fb;
	for (int i = 1; i < itmax; i++)
	{
		if (fb * fc > 0.0)
		{
			c = a;
			fc = fa;
			d = b - a;
			e = d;
		}
		if (abs(fc) < abs(fb))
		{
			a = b;
			b = c;
			c = a;
			fa = fb;
			fb = fc;
			fc = fa;
		}
		tol1 = 2.0 * eps * abs(b) + 0.5 * tol;
		xm = 0.5 * (c - b);
		if ((abs(xm) < tol1) || (fb == 0)) return res;
		if ((abs(e) > tol1) && (abs(fa) > abs(fb)))
		{
			s = fb / fa;
			if (a == c)
			{
				p = 2.0 * xm * s;
				q = 1.0 - s;
			}
			else
			{
				q = fa / fc;
				r = fb / fc;
				p = s * (2.0 * xm * q * (q - r) - (b - a) * (r - 1.0));
				q = (q - 1.0) * (r - 1.0) * (s - 1.0);
			}
			if (p > 0.0) q = -q;
			p = abs(p);
			if (2.0 * p < min(3.0 * xm * q - abs(tol1 * q), abs(e * q)))
			{
				e = d;
				d = p / q;
			}
			else
			{
				d = xm;
				e = d;
			}
		}
		else
		{
			d = xm;
			e = d;
		}
		a = b;
		fa = fb;
		if (abs(d) > tol1) b += d;
		else
		{
			if (xm > 0.0) b = b + abs(tol1);
			else b = b - abs(tol1);
		}
		mi = occ_solve1D_zbr(sc_occ, sxc, syc, tol, b, gamma0, nl, tp);
		fb = f(b, mi, gamma0, Vs, tp);
		res = { b,mi };
	}

	cout << "zbr exeding max iteractions!" << endl;
	return res;
}

Result_Pair singlet_get_res(double Vs, double nl, double tp, double gamma0, double xt, double yt, double xc, double yc)
{
	double ch, tol = 0.000000001;

	Result_Pair res = sc_solve1D_zbr(singlet_gap, xt, yt, tol, gamma0, Vs, xc, yc, nl, tp);
	/* ch = occ_solve1D_zbr(sc_occ, xc, yc, tol, res.T, gamma0, nl, tp);

	 while(abs(ch-res.mi)>tol)
	 {
		 res=sc_solve1D_zbr(singlet_gap,xt,yt,tol,gamma0,Vs,res.mi *.05,res.mi *1.5,nl,tp);
		 ch=occ_solve1D_zbr(sc_occ,xc,yc,tol,res.T,gamma0,nl,tp);
	 }
	 */
	std::cout << "singlet ch " << res.mi << " t " << res.T << std::endl;
	return res;
}

double tryplet_gap(double Tc, double mi, double gamma0, double Vt, double tp)
{
	const double pi = asin(1.0) * 2.0;
	double temp;

	temp = sc_integrate1D_gl_gpu(i_tryplet_uifunt, 0.0, pi, Tc, mi, gamma0, tp);
	temp = 1.0 - Vt / (2.0 * pi) * (temp);
	return temp;

}

Result_Pair tryplet_get_res(double Vt, double nl, double tp, double gamma0, double xt, double yt, double xc, double yc)
{
	double t, ch, chp, tol = 0.000000001;
	Result_Pair	res = sc_solve1D_zbr(tryplet_gap, xt, yt, tol, gamma0, Vt, xc, yc, nl, tp);

	/*cout<<"t tryplet ch "<<chp<<" t "<<t<<endl;
	t=sc_solve1D_zbr(tryplet_gap,xt,yt,tol,0.0,gamma0,n,k,Vt);
	cout<<"ch tryplet ch "<<chp<<" t "<<t<<endl;
	ch=sc_solve1D_zbr(sc_occ,xc,yc,tol,t,gamma0,n,k,nl);

	chp=0.0;

	while(abs(ch-chp)>tol)
	{
		chp=ch;
		cout<<"t tryplet ch "<<chp<<" t "<<t<<endl;
		t=sc_solve1D_zbr(tryplet_gap,xt,yt,tol,chp,gamma0,n,k,Vt);
		cout<<"ch tryplet ch "<<chp<<" t "<<t<<endl;
		ch=sc_solve1D_zbr(sc_occ,xc,yc,tol,t,gamma0,n,k,nl);
	}

		res[0]=t;
		res[1]=ch;*/
	std::cout << "tryplet ch " << res.mi << " t " << res.T << std::endl;

	return res;
}

struct Density_Result {
	double plus;
	double minus;
};

__device__ Density_Result sc_dos(double o, double g0, double tp)
{
	int npi = 100000;
	double ddos = 1.e-02;
	double step = 2.0 * pi_device / npi;
	double densitytot = 2. * npi;
	double omega, gamma, omp, omm;

	double domega = 2. * ddos;

	double densityp = 0.;
	double densitym = 0.;

	double x = -pi_device;
	for (int j = 0; j < npi; j++)
	{
		omega = ek(x, tp);
		gamma = g(x, g0);
		omp = omega + gamma;
		omm = omega - gamma;
		if (abs(omp - o) < ddos) densityp = densityp + 1.;
		if (abs(omm - o) < ddos) densitym = densitym + 1.;
		x = x + step;
	}
	densitym = densitym / (densitytot * domega);
	densityp = densityp / (densitytot * domega);
	return { densityp,densitym };
}

double fs(double gamma0, double mi, double lambda)
{
	double a = 1 + 0.25 * gamma0 * gamma0 - 0.25 * mi * mi;
	if (a > 0)
	{
		a = -0.5 * mi + lambda * 0.5 * gamma0 * sqrt(a);
	}
	else
	{
		a = -0.5 * mi;
	}
	a = a / (1 + 0.25 * gamma0 * gamma0);
	return a;
}


void sc_tabulate1D(string fname, Result_Pair(*f)(double, double, double, double, double, double, double, double), double a, double b, int N, double Vs, double tp, double nl, double xt, double yt, double xc, double yc)
{
	fstream outfile(fname, fstream::out);
	double g0 = a;
	double h = (b - a) / (N - 1);

	if (!outfile.good())
	{
		cout << "nie otwarty plik!" << endl;
	}

	while (h > 0.0001)
	{
		g0 = g0 + h;
		Result_Pair res = f(Vs, nl, tp, g0, xt, yt, xc, yc);
		if (res.T == 0.0)
		{
			g0 = g0 - h;
			h = h / 2.0;
		}
		else
		{
			outfile << setprecision(10) << g0 << " " << setprecision(10) << res.T << " " << setprecision(10) << res.mi << endl;
		}

	}
}

int main()
{
	double xt = 0.000000001, yt = 10.0, xc = 0.0000001, yc = 5.0;
	double tp = 0.0, nl = 1.2;
	double Vs = 1.2;

	sc_tabulate1D("tryplet.txt", tryplet_get_res, 0.01, 0.5, 100, Vs, tp, nl, xt, yt, xc, yc); //fname,f,a,b,N,Vs,tp,nl,xt,yt,xc,yc 
	return 0;
}
