#include <iostream>
#include <fstream>
#include <iomanip>
#include <sstream>
#include <chrono>
#include <numeric>
#include <vector>
#include "cmath"
#include <string>
#include <cmath>
#include <hip/hip_runtime.h>
#include <string>
#include <fstream>

using namespace std;

#define BLOCK_SIZE 16
const double h_pi=3.141592653597932384;
__constant__ double pi_device =3.141592653597932384;
const double pi = 3.141592653597932384;
const double delta = 0.0000000001;
const int n=10000, k=5;

const double xk[]={0.9061798459386641,-0.9061798459386641,0.538469310105683,-0.5384693101056829,0.0};
const double ak[]={0.2369268850561876,0.2369268850561876,0.47862867049936647,0.47862867049936586,0.5688888888888889};

typedef double (*Under_Integral_Func)(double,double,double,double,double);

__device__ double g(double x,double gamma0)
{
    return gamma0*(abs(sin(x)));
}

__device__ double ek(double x,double tp)
{
    return -2.0*(cos(x))+2.0*tp*cos(2*x);
}

__device__ double sc_uifunc(double x,double Tc,double mi,double gamma0,double tp)
{
    double ksip=ek(x,tp)-mi+g(x,gamma0),ksim=ek(x,tp)-mi-g(x,gamma0);

    return tanh(ksip/(2.0*Tc))+tanh(ksim/(2.0*Tc));
}

__device__ double singlet_uifunt(double x,double Tc,double mi,double gamma0,double tp)
{
    double ksip=ek(x,tp)-mi+g(x,gamma0),ksim=ek(x,tp)-mi-g(x,gamma0);
    double res;

    if(abs(ksim)<0.0000000000001)
        res= 1.0/(2.0*Tc);
    else
        res= tanh(ksim/(2.0*Tc))/(2.0*ksim);

    if(abs(ksip)<0.0000000000001)
        res+= 1.0/(2.0*Tc);
    else
        res+= tanh(ksip/(2.0*Tc))/(2.0*ksip);

    return res;
}
__device__ Under_Integral_Func uif_array[] = { sc_uifunc, singlet_uifunt };
enum UI_Func_Index { i_sc_uifunc=0, i_singlet_uifunt=1};

__global__ void integrateKernel(double* result, UI_Func_Index i_f, double a, double h, int n, double* xk, double* ak, double Tc, double mi, double gamma0, double tp) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        int j = idx % 5;
        int i = idx / 5;
        double z = ((2.0*a+h*(2.0*i+1.0))-h*xk[j])*0.5;
        result[idx] = ak[j]*((uif_array[i_f])(z, Tc, mi, gamma0, tp));
    }
}

double sc_integrate1D_gl_gpu(UI_Func_Index i_f, double a, double b, double Tc, double mi, double gamma0, double tp) {
    double h = (b - a) / (n);
   // Host variables
    double* h_result = new double[n * k];

    // Device variables
    double* d_result;
    double* d_xk;
    double* d_ak;
    hipMalloc((void**)&d_result, n * k * sizeof(double));
    hipMalloc((void**)&d_xk, k * sizeof(double));
    hipMalloc((void**)&d_ak, k * sizeof(double));

    // Copy data to device
    hipMemcpy(d_xk, xk, k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_ak, ak, k * sizeof(double), hipMemcpyHostToDevice);

    // Launch the kernel
    int num_blocks = (n*k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    integrateKernel<<<num_blocks, BLOCK_SIZE>>>(d_result, i_f, a, h, n*k, d_xk, d_ak, Tc, mi, gamma0, tp);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed: %s\n", hipGetErrorString(cudaStatus));
    }
    // Copy the result back to host
    hipMemcpy(h_result, d_result, n * k * sizeof(double), hipMemcpyDeviceToHost);
    
    double result = accumulate(h_result, h_result + n*k, 0.0) * h * 0.5;
    // Cleanup
    hipFree(d_result);
    hipFree(d_xk);
    hipFree(d_ak);
    delete[] h_result;
    return result;
}

double singlet_gap(double Tc,double mi,double gamma0,double Vs,double tp)
{
    return 1.0-Vs/(2.0*pi)*sc_integrate1D_gl_gpu(i_singlet_uifunt,0.0,pi,Tc,mi,gamma0,tp);
}

double sc_occ(double mi,double Tc,double gamma0,double nl,double tp)
{
    return 1.0-nl-sc_integrate1D_gl_gpu(i_sc_uifunc,0.0,pi,Tc,mi,gamma0,tp)/(2.0*pi);
}

typedef double (*Gap_Func)(double,double,double,double,double);


double occ_solve1D_zbr(Gap_Func f,double a, double b,double tol,double T,double gamma0,double nl,double tp)
{
    int itmax=100;
    double d,r,s,e,p,q,xm,tol1,c,fa,fb,fc;

    fa=f(a,T,gamma0,nl,tp);
    fb=f(b,T,gamma0,nl,tp);

    if(fa*fb>0.0)
    {
        cout<<"occ_zbr err:Takie same znaki!  fa  "<<fa<<" fb  "<<fb<<" tc,g0,a,b  "<<T<<"  "<<gamma0<<"  "<<a<<"  "<<b<<endl;
        return 0.0;
    }
    c=b;
    fc=fb;
    for(int i=1;i<itmax;i++)
    {
        if(fb*fc>0.0)
        {
            c=a;
            fc=fa;
            d=b-a;
            e=d;
        }
        if(abs(fc)<abs(fb))
        {
            a=b;
            b=c;
            c=a;
            fa=fb;
            fb=fc;
            fc=fa;
        }
        tol1=2.0*delta*abs(b)+0.5*tol;
        xm=0.5*(c-b);
        if((abs(xm)<tol1)||(fb==0)) return b;
        if((abs(e)>tol1)&&(abs(fa)>abs(fb)))
        {
            s=fb/fa;
            if(a==c)
            {
                p=2.0*xm*s;
                q=1.0-s;
            }
            else
            {
                q=fa/fc;
                r=fb/fc;
                p=s*(2.0*xm*q*(q-r)-(b-a)*(r-1.0));
                q=(q-1.0)*(r-1.0)*(s-1.0);
            }
            if(p>0.0) q=-q;
            p=abs(p);
            if(2.0*p<min(3.0*xm*q-abs(tol1*q),abs(e*q)))
            {
                e=d;
                d=p/q;
            }
            else
            {
                d=xm;
                e=d;
            }
        }
        else
        {
            d=xm;
            e=d;
        }
        a=b;
        fa=fb;
        if(abs(d)>tol1) b+=d;
        else
        {
            if(xm>0.0) b=b+abs(tol1);
            else b=b-abs(tol1);
        }
        fb=f(b,T,gamma0,nl,tp);


    }
    cout<<"zbr exeding max iteractions!"<<endl;
    return b;
}

struct Result_Pair {
    double T;
    double mi;
};

Result_Pair sc_solve1D_zbr(Gap_Func f, double a, double b, double tol, double gamma0, double Vs, double sxc, double syc, double nl, double tp)
{
    int itmax=100;
    double d,r,s,e,p,q,xm,tol1,c,fa,fb,fc,eps=3.0e-8,mi;
    mi=occ_solve1D_zbr(sc_occ,sxc,syc,tol,a,gamma0,nl,tp);
    fa=f(a,mi,gamma0,Vs,tp);
    mi=occ_solve1D_zbr(sc_occ,sxc,syc,tol,b,gamma0,nl,tp);
    fb=f(b,mi,gamma0,Vs,tp);
    Result_Pair res = { b, mi };

    if(fa*fb>0.0)
    {
        cout<<"sc_zbr err:Takie same znaki!  fa  "<<fa<<" fb  "<<fb<<" mi,g0,a,b  "<<mi<<"  "<<gamma0<<"  "<<a<<"  "<<b<<endl;
        return { 0.0,0.0 };
    }
    c=b;
    fc=fb;
    for(int i=1;i<itmax;i++)
    {
        if(fb*fc>0.0)
        {
            c=a;
            fc=fa;
            d=b-a;
            e=d;
        }
        if(abs(fc)<abs(fb))
        {
            a=b;
            b=c;
            c=a;
            fa=fb;
            fb=fc;
            fc=fa;
        }
        tol1=2.0*eps*abs(b)+0.5*tol;
        xm=0.5*(c-b);
        if((abs(xm)<tol1)||(fb==0)) return res;
        if((abs(e)>tol1)&&(abs(fa)>abs(fb)))
        {
            s=fb/fa;
            if(a==c)
            {
                p=2.0*xm*s;
                q=1.0-s;
            }
            else
            {
                q=fa/fc;
                r=fb/fc;
                p=s*(2.0*xm*q*(q-r)-(b-a)*(r-1.0));
                q=(q-1.0)*(r-1.0)*(s-1.0);
            }
            if(p>0.0) q=-q;
            p=abs(p);
            if(2.0*p<min(3.0*xm*q-abs(tol1*q),abs(e*q)))
            {
                e=d;
                d=p/q;
            }
            else
            {
                d=xm;
                e=d;
            }
        }
        else
        {
            d=xm;
            e=d;
        }
        a=b;
        fa=fb;
        if(abs(d)>tol1) b+=d;
        else
        {
            if(xm>0.0) b=b+abs(tol1);
            else b=b-abs(tol1);
        }
        mi=occ_solve1D_zbr(sc_occ,sxc,syc,tol,b,gamma0,nl,tp);
        fb=f(b,mi,gamma0,Vs,tp);
        res = { b,mi };
    }

    cout<<"zbr exeding max iteractions!"<<endl;
    return res;
}

Result_Pair singlet_get_res(double Vs,double nl,double tp,double gamma0,double xt,double yt,double xc,double yc)
{
    double ch,tol=0.000000001;
    
    Result_Pair res=sc_solve1D_zbr(singlet_gap,xt,yt,tol,gamma0,Vs,xc,yc,nl,tp);
   /* ch = occ_solve1D_zbr(sc_occ, xc, yc, tol, res.T, gamma0, nl, tp);

    while(abs(ch-res.mi)>tol)
    {
        res=sc_solve1D_zbr(singlet_gap,xt,yt,tol,gamma0,Vs,res.mi *.05,res.mi *1.5,nl,tp);
        ch=occ_solve1D_zbr(sc_occ,xc,yc,tol,res.T,gamma0,nl,tp);
    }
    */
      std::cout<<"singlet ch "<<res.mi <<" t "<<res.T<<std::endl;
      return res;
}

struct Density_Result {
    double plus;
    double minus;
};

__device__ Density_Result sc_dos(double o,double g0,double tp)
{
    int npi=100000;
    double ddos=1.e-02;
	double step=2.0*pi_device/npi;
	double densitytot=2.*npi;
    double omega,gamma,omp,omm;

	double domega=2.*ddos;

    double densityp=0.;
    double densitym=0.;

    double x=-pi_device;
    for(int j=0;j<npi;j++)
    {
      omega=ek(x,tp);
      gamma=g(x,g0);
      omp=omega+gamma;
      omm=omega-gamma;
      if (abs(omp-o) < ddos) densityp=densityp+1.;
      if (abs(omm-o) < ddos) densitym=densitym+1.;
      x=x+step;
    }
    densitym=densitym/(densitytot*domega);
    densityp=densityp/(densitytot*domega);
    return {densityp,densitym};
}

double fs(double gamma0,double mi,double lambda)
{
    double a=1+0.25*gamma0*gamma0-0.25*mi*mi;
    if(a>0)
    {
        a=-0.5*mi+lambda*0.5*gamma0*sqrt(a);
    }
    else
    {
        a=-0.5*mi;
    }
    a=a/(1+0.25*gamma0*gamma0);
    return a;
}


void sc_tabulate1D(string fname, Result_Pair (*f)(double,double,double,double,double,double,double,double),double a, double b, int N,double Vs,double tp,double nl,double xt,double yt,double xc,double yc)
{
    fstream outfile(fname,fstream::out);
    double g0=a;
    double h=(b-a)/(N-1);

    if(!outfile.good())
    {
        cout<<"nie otwarty plik!"<<endl;
    }

    while(h>0.0001)
    {
        g0=g0+h;
        Result_Pair res = f(Vs,nl,tp,g0,xt,yt,xc,yc);
        if(res.T == 0.0)
        {
            g0=g0-h;
            h=h/2.0;
        }
        else
        {
            outfile<<setprecision(10)<<g0<<" "<<setprecision(10)<<res.T<<" "<<setprecision(10)<<res.mi<<endl;
        }

    }
  }

int main()
{
    double xt=0.000000001,yt=10.0,xc=0.0000001,yc=5.0;
    double tp=0.0,nl=1.2,ch,mi,dmi;
    double Vs = 0.6;
    string fname= "singlet.txt";

    //singlet_get_res(1.0,nl,tp,g0,xt,yt,xc,yc,100,5); //Vs,nl,tp,g0,xt,yt,xc,yc,n,k
    sc_tabulate1D(fname,singlet_get_res,0.01,0.5,100,Vs,tp,nl,xt,yt,xc,yc); //fname,f,a,b,N,Vs,tp,nl,xt,yt,xc,yc 
    return 0;
}
